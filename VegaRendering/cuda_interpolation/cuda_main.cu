#include "hip/hip_runtime.h"  
#include "hipblas.h"  

#include <time.h>  
#include <iostream>  

using namespace std;

// ������Ծ����ά��  
int const M = 5;
int const N = 10;

int main()
{
	// ����״̬����  
	hipblasStatus_t status;

	// �� �ڴ� ��Ϊ��Ҫ����ľ��󿪱ٿռ�  
	float *h_A = (float*)malloc(N*M * sizeof(float));
	float *h_B = (float*)malloc(N*M * sizeof(float));

	// �� �ڴ� ��Ϊ��Ҫ����������ľ��󿪱ٿռ�  
	float *h_C = (float*)malloc(M*M * sizeof(float));

	// Ϊ����������Ԫ�ظ��� 0-10 ��Χ�ڵ������  
	for (int i = 0; i < N*M; i++) {
		h_A[i] = (float)(rand() % 10 + 1);
		h_B[i] = (float)(rand() % 10 + 1);

	}

	// ��ӡ�����Եľ���  
	cout << "���� A :" << endl;
	for (int i = 0; i < N*M; i++) {
		cout << h_A[i] << " ";
		if ((i + 1) % N == 0) cout << endl;
	}
	cout << endl;
	cout << "���� B :" << endl;
	for (int i = 0; i < N*M; i++) {
		cout << h_B[i] << " ";
		if ((i + 1) % M == 0) cout << endl;
	}
	cout << endl;

	/*
	** GPU ����������
	*/

	// ��������ʼ�� CUBLAS �����  
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);

	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
			cout << "CUBLAS ����ʵ��������" << endl;
		}
		getchar();
		return EXIT_FAILURE;
	}

	float *d_A, *d_B, *d_C;
	// �� �Դ� ��Ϊ��Ҫ����ľ��󿪱ٿռ�  
	hipMalloc(
		(void**)&d_A,    // ָ�򿪱ٵĿռ��ָ��  
		N*M * sizeof(float)    //����Ҫ���ٿռ���ֽ���  
	);
	hipMalloc(
		(void**)&d_B,
		N*M * sizeof(float)
	);

	// �� �Դ� ��Ϊ��Ҫ����������ľ��󿪱ٿռ�  
	hipMalloc(
		(void**)&d_C,
		M*M * sizeof(float)
	);

	// ���������ݴ��ݽ� �Դ� ���Ѿ����ٺ��˵Ŀռ�  
	hipblasSetVector(
		N*M,    // Ҫ�����Դ��Ԫ�ظ���  
		sizeof(float),    // ÿ��Ԫ�ش�С  
		h_A,    // ��������ʼ��ַ  
		1,    // ����Ԫ��֮��Ĵ洢���  
		d_A,    // GPU ����ʼ��ַ  
		1    // ����Ԫ��֮��Ĵ洢���  
	);
	hipblasSetVector(
		N*M,
		sizeof(float),
		h_B,
		1,
		d_B,
		1
	);

	// ͬ������  
	hipDeviceSynchronize();

	// ���ݽ�������˺����еĲ��������庬����ο������ֲᡣ  
	float a = 1; float b = 0;
	// ������ˡ��ú�����Ȼ���������������������  
	hipblasSgemm(
		handle,    // blas �����   
		HIPBLAS_OP_T,    // ���� A ���Բ���  
		HIPBLAS_OP_T,    // ���� B ���Բ���  
		M,    // A, C ������   
		M,    // B, C ������  
		N,    // A �������� B ������  
		&a,    // ����ʽ�� �� ֵ  
		d_A,    // A ���Դ��еĵ�ַ  
		N,    // lda  
		d_B,    // B ���Դ��еĵ�ַ  
		M,    // ldb  
		&b,    // ����ʽ�� �� ֵ  
		d_C,    // C ���Դ��еĵ�ַ(�������)  
		M    // ldc  
	);

	// ͬ������  
	hipDeviceSynchronize();

	// �� �Դ� ��ȡ���������� �ڴ���ȥ  
	hipblasGetVector(
		M*M,    //  Ҫȡ��Ԫ�صĸ���  
		sizeof(float),    // ÿ��Ԫ�ش�С  
		d_C,    // GPU ����ʼ��ַ  
		1,    // ����Ԫ��֮��Ĵ洢���  
		h_C,    // ��������ʼ��ַ  
		1    // ����Ԫ��֮��Ĵ洢���  
	);

	// ��ӡ������  
	cout << "��������ת�� ( (A*B)��ת�� )��" << endl;

	for (int i = 0; i < M*M; i++) {
		cout << h_C[i] << " ";
		if ((i + 1) % M == 0) cout << endl;
	}

	// �����ʹ�ù����ڴ�  
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// �ͷ� CUBLAS �����  
	hipblasDestroy(handle);

	getchar();

	return 0;
}